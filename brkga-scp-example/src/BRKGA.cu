#include "hip/hip_runtime.h"
/*
 *
 *  Created on: 2019
 *      Author: Eduardo Xavier
 *
 *
 */


#include "BRKGA.h"


/***
Constructor
***/
BRKGA::BRKGA(unsigned n, unsigned p, float pe, float pm, float rhoe, unsigned K, unsigned decode_type, unsigned NUM_THREADS, unsigned RAND_SEED){
	if(p%THREADS_PER_BLOCK != 0){
			//round population size to a multiple of THREADS_PER_BLOCK
		std::cout<<"Population size rounded from " << p;
		p = ((p/THREADS_PER_BLOCK)+1)*THREADS_PER_BLOCK;
		std::cout<<" to "<< p << std::endl;
	}

	//set to the maximum number of blocks allowed in CUDA compute capability 2.0
	if(K > (unsigned)(2<<30)){
		K = (unsigned) 2<<30;
	}

	this->population_size = p;
	this->number_populations = K;
	this->number_chromosomes = p * K; 
	this->chromosome_size = n;
	this->elite_size = (unsigned)(pe*p);
	this->mutants_size = (unsigned)(pm*p);
	this->rhoe = rhoe;
	this->decode_type = decode_type;
	this->NUM_THREADS = NUM_THREADS;

	using std::range_error;
	if(chromosome_size == 0) { throw range_error("Chromosome size equals zero."); }
	if(population_size == 0) { throw range_error("Population size equals zero."); }
	if(elite_size == 0) { throw range_error("Elite-set size equals zero."); }
	if(elite_size + mutants_size > population_size) { throw range_error("elite + mutant sets greater than population size (p)."); }
	if(number_populations == 0) { throw range_error("Number of parallel populations cannot be zero."); }

	long unsigned total_memory=0;
	// Allocate a float array representing all K populations on host and device
	h_population = (float *)malloc(number_chromosomes*chromosome_size*sizeof(float));
	total_memory += number_chromosomes*chromosome_size*sizeof(float);
	test_memory_malloc(hipMalloc((void **)&d_population, number_chromosomes*chromosome_size*sizeof(float)), 1, total_memory);

	total_memory += number_chromosomes*chromosome_size*sizeof(float);
	test_memory_malloc(hipMalloc((void **)&d_population2, number_chromosomes*chromosome_size*sizeof(float)), 2, total_memory);

	total_memory += number_chromosomes*sizeof(float);
	   // Allocate an array representing the scores of each chromosome on host and device
	h_scores = (float *)malloc(number_chromosomes*sizeof(float));
	test_memory_malloc(hipMalloc((void **)&d_scores, number_chromosomes*sizeof(float)), 3, total_memory);

	total_memory += number_chromosomes*sizeof(PopIdxThreadIdxPair);
	   // Allocate an array representing the indices of each chromosome on host and device
	h_scores_idx = (PopIdxThreadIdxPair *)malloc(number_chromosomes*sizeof(PopIdxThreadIdxPair));
	test_memory_malloc(hipMalloc((void **)&d_scores_idx, number_chromosomes*sizeof(PopIdxThreadIdxPair)), 4, total_memory);

	total_memory += number_chromosomes*chromosome_size*sizeof(ChromosomeGeneIdxPair);
	   // Allocate an array representing the indices of each gene of each chromosome on host and device
	h_chromosome_gene_idx = (ChromosomeGeneIdxPair *)malloc(number_chromosomes*chromosome_size*sizeof(ChromosomeGeneIdxPair));
	test_memory_malloc(hipMalloc((void **)&d_chromosome_gene_idx, number_chromosomes*chromosome_size*sizeof(ChromosomeGeneIdxPair)), 5, total_memory);

	total_memory += number_chromosomes*sizeof(float);
	test_memory_malloc(hipMalloc((void **)&d_random_elite_parent, number_chromosomes*sizeof(float)), 6, total_memory);

	total_memory += number_chromosomes*sizeof(float);
	test_memory_malloc(hipMalloc((void **)&d_random_parent, number_chromosomes*sizeof(float)), 7, total_memory);

	// Allocate a poll to save the POOL_SIZE best solutions, where the first value in each chromosome is the chromosome score
	h_best_solutions = (float *)malloc(POOL_SIZE*(chromosome_size+1)*sizeof(float));
	test_memory_malloc(hipMalloc((void **)&d_best_solutions, POOL_SIZE*(chromosome_size+1)*sizeof(float)), 8, total_memory);

	printf("Total Memory Used In GPU %lu bytes(%lu Mbytes)\n", total_memory, total_memory/1000000);

	this->dimBlock.x = THREADS_PER_BLOCK;
	this->dimGrid.x = (population_size*number_populations)/THREADS_PER_BLOCK;


	// Create pseudo-random number generator 
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	// Set seed 
	hiprandSetPseudoRandomGeneratorSeed(gen, RAND_SEED);
	//Initialize population with random alleles with generated random floats on device 
	reset_population();

}

BRKGA::~BRKGA(){
	// Cleanup 
	hiprandDestroyGenerator(gen);

	hipFree(d_population);
	hipFree(d_population2);
	free(h_population);  

	hipFree(d_scores);     
	free(h_scores);

	hipFree(d_scores_idx);
	free(h_scores_idx);

	hipFree(d_chromosome_gene_idx);
	free(h_chromosome_gene_idx);

	hipFree(d_random_elite_parent);
	hipFree(d_random_parent);

	hipFree(d_best_solutions);
	free(h_best_solutions);


	if(d_instance_info != NULL){
		hipFree(d_instance_info);
		d_instance_info = NULL;
	}
}


void BRKGA::test_memory_malloc(hipError_t err, unsigned code, unsigned total_memory){
	if(err != hipSuccess){
		fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
		fprintf(stderr, "In hipMalloc: %u with total memory %u\n", code, total_memory);
		exit(1);
	}
}


/***
	Allocate information used to evaluate chromosomes on the device.
	It also receives the number of elements (num) in the array info and the size (size) of each element.
	Notice we assume the type of the info elements to be float.
***/
void BRKGA::setInstanceInfo(void *info, long unsigned num, long unsigned size){
	if(info != NULL){
		long unsigned total_memory = num*size;
		printf("Extra Memory Used In GPU due to Instance Info %lu bytes(%lu Mbytes)\n", total_memory, total_memory/1000000);

		if(decode_type == DEVICE_DECODE || decode_type == DEVICE_DECODE_CHROMOSOME_SORTED){
			test_memory_malloc(hipMalloc((void **)&d_instance_info, num*size),8,total_memory);
			hipMemcpy(d_instance_info, info, num*size, hipMemcpyHostToDevice);
		}
		h_instance_info = info;
	}
}


/***
	Generate random alleles for all chromosomes on GPGPU.
***/
void BRKGA::reset_population(void){
	hiprandGenerateUniform(gen, d_population, number_chromosomes*chromosome_size);
}


/***
	If HOST_DECODE is used then this function decodes each cromosome with the host_decode function
	provided in Decoder.cpp
***/
void BRKGA::evaluate_chromosomes_host(){
	hipMemcpy(h_population, d_population, number_chromosomes*chromosome_size*sizeof(float),hipMemcpyDeviceToHost);

	#pragma omp parallel for default(none) shared(dimGrid,dimBlock,h_population,h_scores) collapse(2) num_threads(NUM_THREADS)
	for(int b=0; b < dimGrid.x ; b++){
		for(int t=0; t < dimBlock.x; t++){
			unsigned tx = b*dimBlock.x + t; //Found the thread index since each thread is associated with
			                                //a cromosome.
			float *chromosome = h_population + (tx*chromosome_size);
			h_scores[tx] = host_decode(chromosome, chromosome_size, h_instance_info);
		}
	}	
	hipMemcpy(d_scores, h_scores, number_chromosomes*sizeof(float),hipMemcpyHostToDevice);
}



/***
	If DEVICE_DECODE is used then this kernel function decodes each cromosome with the device_decode function
	provided in Decoder.cpp.
	We use one thread per cromosome to process them.
***/
__global__ 
void decode(float *d_scores, float *d_population, int chromosome_size, void * d_instance_info){
	unsigned global_tx = blockIdx.x*blockDim.x + threadIdx.x;	
	d_scores[global_tx] = device_decode(d_population + global_tx*chromosome_size, chromosome_size, d_instance_info);
}

/***
	If DEVICE_DECODE is used then this function decodes each cromosome with the kernel function
	decode above.
***/
void BRKGA::evaluate_chromosomes_device(){
	//Make a copy of chromossomes to d_population2 such that they can be messed up inside
	//the decoder functions without afecting the real chromosomes on d_population.
	hipMemcpy(d_population2, d_population, number_chromosomes*chromosome_size*sizeof(float),hipMemcpyDeviceToDevice);
	decode<<<dimGrid, dimBlock>>>(d_scores, d_population2, chromosome_size, d_instance_info);
}




/***
	If DEVICE_DECODE_CHROMOSOME_SORTED is used then this kernel function decodes each cromosome with the
	device_decode_chromosomeSorted function	provided in Decoder.cpp.
	We use one thread per cromosome to process them.

	Notice that we use the struct ChromosomeGeneIdxPair since the cromosome is given already sorted to
	the function, and so it has a field with the original index of each gene in the cromosome.
***/
__global__ 
void decode_chromosomes_sorted(float *d_scores, ChromosomeGeneIdxPair *d_chromosome_gene_idx, int chromosome_size, void *d_instance_info){
	unsigned global_tx = blockIdx.x*blockDim.x + threadIdx.x;	
	d_scores[global_tx] = device_decode_chromosome_sorted(d_chromosome_gene_idx + global_tx*chromosome_size, chromosome_size, d_instance_info);
}

/***
	If DEVICE_DECODE_CHROMOSOME_SORTED is used then this function decodes each cromosome with the kernel function
	decode_chromosomes_sorted above. But first we sort each chromosome by its genes values. We save this information
	in the struct ChromosomeGeneIdxPair d_chromosome_gene_idx.
***/
void BRKGA::evaluate_chromosomes_sorted_device(){
	sort_chromosomes_genes();
	decode_chromosomes_sorted<<<dimGrid, dimBlock>>>(d_scores, d_chromosome_gene_idx, chromosome_size, d_instance_info);
}

/***
 If DEVICE_DECODE_CHROMOSOME_SORTED is used.
 Kernel function used to save for each gene of each chromosome, the chromosome index, and the original gene index.
 Used later to sort all chromossomes by gene values. We save gene indexes to preserve this information after sorting.
***/
__global__ 
void device_set_chromosome_gene_idx(ChromosomeGeneIdxPair *d_chromosome_gene_idx, int chromosome_size){
	int tx = blockIdx.x*blockDim.x + threadIdx.x;
	for(int i=0; i<chromosome_size; i++){
		d_chromosome_gene_idx[tx*chromosome_size + i].chromosomeIdx = tx;
		d_chromosome_gene_idx[tx*chromosome_size + i].geneIdx = i;		
	} 
}

/***
 If DEVICE_DECODE_CHROMOSOME_SORTED is used.
 Used as comparator to sort genes of the chromosomes. 
 After sorting by gene we need to reagroup genes by their chromosomes which are indexed by threadIdx.
***/
__device__ bool operator<(const ChromosomeGeneIdxPair& lhs, const ChromosomeGeneIdxPair& rhs){
	return lhs.chromosomeIdx < rhs.chromosomeIdx;
}

/***
  If DEVICE_DECODE_CHROMOSOME_SORTED is used.
	We sort the genes of each chromosome.
	We perform 2 stable_sort sorts: first we sort all genes of all chromosomes by their values, and than we sort by the chromosome index,
	and since stable_sort is used, for each chromosome we will have its genes sorted by their values.
***/
void BRKGA::sort_chromosomes_genes(){
	//First set for each gene, its chromosome index and its original index in the chromosome
	device_set_chromosome_gene_idx<<<dimGrid, dimBlock>>>(d_chromosome_gene_idx, chromosome_size);
  //we use d_population2 to sorte all genes by their values
	hipMemcpy(d_population2, d_population, number_chromosomes*chromosome_size*sizeof(float), hipMemcpyDeviceToDevice);
	
	thrust::device_ptr<float> keys(d_population2);
	thrust::device_ptr<ChromosomeGeneIdxPair> vals(d_chromosome_gene_idx);
	//stable sort both d_population2 and d_chromosome_gene_idx by all the genes values
	thrust::stable_sort_by_key(keys, keys + number_chromosomes*chromosome_size, vals);
	//stable sort both d_population2 and d_chromosome_gene_idx by the chromosome index values
	thrust::stable_sort_by_key(vals, vals + number_chromosomes*chromosome_size, keys);
}



/***
Kernel function, where each thread process one chromosome. It receives the current population *d_population, the next population
pointer *d_population2, two random vectors for indices of parents, d_random_elite_parent and d_random_parent,
***/
__global__ 
void device_next_population(float *d_population, float *d_population2, 
	float *d_random_elite_parent, float *d_random_parent, int chromosome_size, 
	unsigned population_size, unsigned elite_size, unsigned mutants_size, float rhoe, PopIdxThreadIdxPair *d_scores_idx){

	unsigned tx = blockIdx.x*blockDim.x + threadIdx.x; //global thread index
	unsigned chromosome_idx = tx*chromosome_size;
	unsigned pop_idx = (unsigned)tx/population_size; //the population index of this thread
	unsigned inside_pop_idx = tx%population_size; 
	//below are the inside population random indexes of a elite parent and regular parent for crossover
	unsigned parent_elite_idx = (unsigned)(ceilf(d_random_elite_parent[tx]*elite_size)-1);
	unsigned parent_idx = (unsigned)(elite_size+ceilf(d_random_parent[tx]*(population_size-elite_size))-1);

	//if inside_pop_idx < elite_size then thread is elite, so we copy elite chromosome to the next population
	if(inside_pop_idx < elite_size){
		unsigned elite_chromosome_idx = d_scores_idx[tx].thIdx*chromosome_size;
		for(int i=0; i<chromosome_size; i++)
			d_population2[chromosome_idx + i] = d_population[elite_chromosome_idx + i];
	}else if(inside_pop_idx < population_size - mutants_size){
		//if inside_pop_idex >= elite_size and inside < population_size - mutants_size
		//thread is responsible to crossover
		unsigned elite_chromosome_idx = d_scores_idx[pop_idx*population_size + parent_elite_idx].thIdx*chromosome_size;
		unsigned parent_chromosome_idx = d_scores_idx[pop_idx*population_size + parent_idx].thIdx*chromosome_size;
		for(int i=0; i<chromosome_size; i++){
			if(d_population2[chromosome_idx + i] <= rhoe)
				//copy allele from elite parent
				d_population2[chromosome_idx + i] = d_population[elite_chromosome_idx + i];
			else
				//copy allele from regular parent
				d_population2[chromosome_idx + i] = d_population[parent_chromosome_idx + i];
		}
	}//in the else case the thread corresponds to a mutant and nothing is done.	
}


/***
Main function of the BRKGA algorithm. It evolves K populations for a certain number of generations.
***/
void BRKGA::evolve(int number_generations){
	using std::domain_error;

	if(decode_type == DEVICE_DECODE){
		evaluate_chromosomes_device();
	}else if(decode_type == DEVICE_DECODE_CHROMOSOME_SORTED){
		evaluate_chromosomes_sorted_device();
	}else if(decode_type == HOST_DECODE){
		evaluate_chromosomes_host();
	}else{
		throw domain_error("Function decode type is unknown");
	}

	//After this call the vector d_scores_idx has all threads sorted by population, and
	//inside each population, threads are sorted by score
	sort_chromosomes();

	//This call initialize the whole area of the next population d_population2 with random values.
	//So mutantes are already build. For the non mutants we use the 
	//random values generated here to perform the crossover on the current population d_population.
	initialize_population(2);

	//generate random numbers to index parents used for crossover
	hiprandGenerateUniform(gen, d_random_elite_parent, number_chromosomes);
	hiprandGenerateUniform(gen, d_random_parent, number_chromosomes);

	//Kernel function, where each thread process one chromosome of the next population.
	device_next_population<<<dimGrid, dimBlock>>>(d_population, d_population2,  d_random_elite_parent,
		d_random_parent, chromosome_size, population_size, elite_size, mutants_size, rhoe, d_scores_idx);

	float *aux = d_population2;
	d_population2 = d_population;
	d_population = aux;
}


void BRKGA::initialize_population(int p){
	if(p==1)
		hiprandGenerateUniform(gen, d_population, number_chromosomes*chromosome_size);
	if(p==2)
		hiprandGenerateUniform(gen, d_population2, number_chromosomes*chromosome_size);
}



/***
Kernel function that sets for each cromosome its global index (among all populations) and its population index.
***/
__global__ 
void device_set_idx(PopIdxThreadIdxPair *d_scores_idx, int population_size){
	int tx = blockIdx.x*blockDim.x + threadIdx.x;
	d_scores_idx[tx].popIdx = tx/population_size; 
	d_scores_idx[tx].thIdx = tx; 

}

/***
Function used to sort chromosomes by population index
***/
__device__ bool operator<(const PopIdxThreadIdxPair& lhs, const PopIdxThreadIdxPair& rhs){
	return lhs.popIdx < rhs.popIdx;
}

/***
 We sort chromosomes for each population.
 We use the thread index to index each population, and perform 2 stable_sort sorts: first we sort by the chromosome scores, 
 and than by their population index, and since stable_sort is used in each population the chromosomes are sorted by scores.
***/
void BRKGA::sort_chromosomes(){
	//For each thread we store in d_scores_idx the global chromosome index and its population index.
	device_set_idx<<<dimGrid, dimBlock>>>(d_scores_idx, population_size);

	thrust::device_ptr<float> keys(d_scores);
	thrust::device_ptr<PopIdxThreadIdxPair> vals(d_scores_idx);
	//now sort all chromosomes by their scores (vals)
	thrust::stable_sort_by_key(keys, keys + number_chromosomes, vals);
	//now sort all chromossomes by their population index
	//in the sorting process it is used operator< above to compare two structs of this type
	thrust::stable_sort_by_key(vals, vals + number_chromosomes, keys);
}





/***
	Kernel function to operate the exchange of elite chromosomes.
	It was launched M*number_populations threads.
	For each population each one of M threads do the copy of an elite chromosome of its own population
	into the other populations.
	To do: make kernel save in local memory the chromosome and then copy to each other population
***/
__global__ 
void device_exchange_elite(float *d_population,  int chromosome_size, unsigned population_size, unsigned number_populations, PopIdxThreadIdxPair *d_scores_idx, unsigned M){

	unsigned tx = threadIdx.x; //this thread value between 0 and M-1
	unsigned pop_idx = blockIdx.x; //this thread population index, a value between 0 and number_populations-1
	unsigned elite_idx = pop_idx*population_size + tx;
	unsigned elite_chromosome_idx = d_scores_idx[elite_idx].thIdx;
	unsigned inside_destiny_idx = population_size-1-(M*pop_idx)-tx;//index of the destiny of this thread inside each population

	for(int i=0; i<number_populations; i++){
		if(i != pop_idx){
			unsigned destiny_chromosome_idx = d_scores_idx[i*population_size + inside_destiny_idx].thIdx;
			for(int j=0; j<chromosome_size;j++)
				d_population[destiny_chromosome_idx*chromosome_size + j] = d_population[elite_chromosome_idx*chromosome_size + j];
		}
	}
}

/***
Exchange M individuals among the different populations.
***/
void BRKGA::exchangeElite(unsigned M){
	using std::range_error;
	if(M > elite_size) { throw range_error("Exchange elite size M greater than elite size."); }
	if(M*number_populations > population_size) { throw range_error("Total exchange elite size greater than population size."); }

	using std::domain_error;
	if(decode_type == DEVICE_DECODE){
		evaluate_chromosomes_device();
	}else if(decode_type == DEVICE_DECODE_CHROMOSOME_SORTED){
		evaluate_chromosomes_sorted_device();
	}else if(decode_type == HOST_DECODE){
		evaluate_chromosomes_host();
	}
	else{
		throw domain_error("Function decode type is unknown");
	}

	sort_chromosomes();
	device_exchange_elite<<<number_populations, M>>>(d_population, chromosome_size, population_size, number_populations, d_scores_idx,  M);
}


/***
	Return a vector of vectors, where each line vector corresponds to a chromosome,
	where in position 0 we have its score and in positions 1 to chromosome_size the aleles values
***/
std::vector<std::vector <float>> BRKGA::getkBestChromosomes(unsigned k){
	std::vector<std::vector <float>> ret(k, std::vector<float>(chromosome_size+1));

	global_sort_chromosomes();
	hipMemcpy(h_scores_idx, d_scores_idx, number_chromosomes*sizeof(PopIdxThreadIdxPair),hipMemcpyDeviceToHost);
	hipMemcpy(h_scores, d_scores, number_chromosomes*sizeof(float),hipMemcpyDeviceToHost);
	hipMemcpy(h_population, d_population, number_chromosomes*chromosome_size*sizeof(float),hipMemcpyDeviceToHost);

	for(int i=0; i<k; i++){
		unsigned tx = h_scores_idx[i].thIdx;
		float *begin = &h_population[tx*chromosome_size];
		ret[i][0] = h_scores[i];
		for(int u=1; u <= chromosome_size; u++){
			ret[i][u] = begin[u-1];
		}
	}
	return ret;
}

/***
  Return a vector of vectors, where each line vector corresponds to a chromosome,
  where in position 0 we have its score and in positions 1 to chromosome_size the aleles values
***/
std::vector<std::vector <float>> BRKGA::getkBestChromosomes2(unsigned k){
	if(k>POOL_SIZE) k=POOL_SIZE;
	std::vector<std::vector <float>> ret(k, std::vector<float>(chromosome_size+1));
	saveBestChromosomes();
	hipMemcpy(h_best_solutions, d_best_solutions, POOL_SIZE*(chromosome_size+1)*sizeof(float),hipMemcpyDeviceToHost);

	for(int i=0; i<k; i++){
		for(int j=0; j <= chromosome_size; j++){
			ret[i][j] = h_best_solutions[i*(chromosome_size+1) + j];
		}
	}

	return ret;
}


__global__ 
void device_save_best_chromosomes(float *d_population,  unsigned chromosome_size,  PopIdxThreadIdxPair *d_scores_idx, float *d_best_solutions, float *d_scores, unsigned best_saved){
	if(!best_saved){//this is the first time saving best solutions in to the pool
		for(int i=0; i<POOL_SIZE; i++){
			unsigned tx = d_scores_idx[i].thIdx;
			float *begin = (float *)&d_population[tx*chromosome_size];
			d_best_solutions[i*(chromosome_size+1)] = d_scores[i]; //save the value of the chromosome
			for(int j=1; j <= chromosome_size; j++){ //save the chromosome
				d_best_solutions[i*(chromosome_size+1)+j] = begin[j-1];
			}
		}
	}else{//Since best solutions were already saved
				//only save now if the i-th best current solution is better than the i-th best overall
		for(int i=0; i<POOL_SIZE; i++){
			unsigned tx = d_scores_idx[i].thIdx;
			float *begin = (float *)&d_population[tx*chromosome_size];
			if(d_scores[i] < d_best_solutions[i*(chromosome_size+1)]){
				d_best_solutions[i*(chromosome_size+1)] = d_scores[i];
				for(int j=1; j <= chromosome_size; j++){
					d_best_solutions[i*(chromosome_size+1)+j] = begin[j-1];
				}
			}
		}
	}
}

/***
 This Function saves in the pool d_best_solutions and h_best_solutions the best solutions generated so far among all populations.
***/
void BRKGA::saveBestChromosomes(){
	global_sort_chromosomes();
	device_save_best_chromosomes<<<1, 1>>>(d_population, chromosome_size, d_scores_idx, d_best_solutions, d_scores, best_saved);
	best_saved = 1;
}

/***
	We sort all chromosomes of all populations toguether.
	We use the global thread index to index each chromosome, since each thread is responsible for one thread.
	Notice that in this function we only perform one sort, since we want the best chromosomes overall, so we do not
	perform a second sort to separate chromosomes by their population.
***/
void BRKGA::global_sort_chromosomes(){
	using std::domain_error;
	if(decode_type == DEVICE_DECODE){
		evaluate_chromosomes_device();
	}else if(decode_type == DEVICE_DECODE_CHROMOSOME_SORTED){
		evaluate_chromosomes_sorted_device();
	}else if(decode_type == HOST_DECODE){
		evaluate_chromosomes_host();
	}
	else{
		throw domain_error("Function decode type is unknown");
	}


	device_set_idx<<<dimGrid, dimBlock>>>(d_scores_idx, population_size);
	thrust::device_ptr<float> keys(d_scores);
	thrust::device_ptr<PopIdxThreadIdxPair> vals(d_scores_idx);
	thrust::sort_by_key(keys, keys + number_chromosomes, vals);
}


